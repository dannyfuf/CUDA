#include <time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <random>
#define BS 256
#define M 10000
#define N 10000
using namespace std;

//j fila
//i columna


void preguntaUno(double *arr, float dx){
    double first, prev, next;
    for(int t = 1; t <= 10; t++){ //itera el t 
        for(int i = 0; i < N*M; i++){ //recorre matriz
            if(i%N == 0 ){
                first = arr[i];
                //es el primero de la fila = primera columna
                prev = (arr[i+1] - arr[i+N-1])/(2*dx);
            }
            else if (i%N == N-1){
                //es el ultimo de la fila = ultima columna
                arr[i] = (first - arr[i-1])/(2*dx);
                arr[i-1] = prev;
            }
            else{
                next = (arr[i+1] - arr[i-1])/(2*dx);
                arr[i-1] = prev;
                prev = next;
            }
        }
    }
}

__global__ void preguntaDos(double *arr, float dx){
    double first, prev, next;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < M){
        for(int t = 1; t <= 10; t++){ //itera el t 
            for(int i = 0; i < N; i++){ //recorre matriz
                if(i == 0){
                    first = arr[i+tid*M];
                    //es el primero de la fila = primera columna
                    prev = (arr[i+1+tid*M] - arr[i+N-1+tid*M])/(2*dx);
                }
                else if (i == N-1){
                    //es el ultimo de la fila = ultima columna
                    arr[i+tid*M] = (first - arr[i-1+tid*M])/(2*dx);
                    arr[i-1+tid*M] = prev;
                }
                else{
                    next = (arr[i+1+tid*M] - arr[i-1+tid*M])/(2*dx);
                    arr[i-1+tid*M] = prev;
                    prev = next;
                }
            }
        }
	}
}

__global__ void preguntaTres(double *arr, float dx){
        printf("------------------------------------------\n");
    double first, prev, next;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < M/4){
        for(int t = 1; t <= 10; t++){ //itera el t 
            for(int i = 0; i < N; i++){ //recorre matriz
                if(i == 0){
                    first = arr[i+tid*M];
                    //es el primero de la fila = primera columna
                    prev = (arr[i+1+tid*M] - arr[i+N-1+tid*M])/(2*dx);
                }
                else if (i == N-1){
                    //es el ultimo de la fila = ultima columna
                    arr[i+tid*M] = (first - arr[i-1+tid*M])/(2*dx);
                    arr[i-1+tid*M] = prev;
                }
                else{
                    next = (arr[i+1+tid*M] - arr[i-1+tid*M])/(2*dx);
                    arr[i-1+tid*M] = prev;
                    prev = next;
                }
            }
        }
	}
    printf("------------------------------------------\n");
}

double * createCopy(double * arr){
    double * newArr = new double[N*M];
    for(int i = 0; i < N*M; i++){
        newArr[i] = arr[i];
    }
    return newArr;
}

int main(){
    //inicializacion del arreglo
    random_device rd;
    mt19937 gen(rd()); 
    uniform_real_distribution<> dis(0.0, 1.0);
    float dx = 0.001;
    double *arr = new double[N*M];
    for(int i = 0; i < N*M; i++){
        arr[i] = dis(gen);
    }

    for(int i = 0; i < 20; i++){
        cout << arr[i] << " ";
    }
    cout << endl;

    //pregunta uno
    // double *arrP1 = createCopy(arr);
    // clock_t t1, t2;
    // int t = 10;
    // double ms;
    // t1 = clock();
    // preguntaUno(arrP1, dx);
    // t2 = clock();
    // ms = 1000.0 * (double)(t2 - t1) / CLOCKS_PER_SEC;
    // cout << "1. Tiempo empleado es: "<< ms << "[ms]" << endl;
    // for(int i = 0; i < 20; i++){
    //     cout << arrP1[i] << " ";
    // }
    // cout << endl;

    //pregunta dos
    double *arrP2 = createCopy(arr);
    hipEvent_t ct1, ct2;
    double *arrCUDA;
    float dt;
    int gs = (int)ceil((float)M / BS);
    hipMalloc((void **)&arrCUDA, N * M * sizeof(double));
    hipMemcpy(arrCUDA, arrP2, N * M * sizeof(double), hipMemcpyHostToDevice);

    hipEventCreate(&ct1);
    hipEventCreate(&ct2);
    hipEventRecord(ct1);
    preguntaDos<<<gs, BS>>>(arrCUDA, dx);
    hipEventRecord(ct2);
    hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt, ct1, ct2);
	printf("2. Tiempo GPU: %f[ms]\n", dt);
    hipMemcpy(arrP2, arrCUDA, N * M * sizeof(double), hipMemcpyDeviceToHost);
    // for(int i = 0; i < 20; i++){
    //    cout << arrP2[i] << " ";
    // }
    // cout << endl;

    //pregunta tres
    double *arrP3, *arrCUDA1, *arrCUDA2, *arrCUDA3, *arrCUDA4;
    hipHostMalloc(&arrP3, M*N*sizeof(double), hipHostMallocDefault);

    for(int i = 0; i < N*M; i++){
        arrP3[i] = arr[i];
    }
    
    hipStream_t stream1, stream2, stream3, stream4;
    gs = (int)ceil((float)(M/4) / BS);
    
    //cudaStream_t stream[4];
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    hipStreamCreate(&stream4);
    
    hipMalloc((void **)&arrCUDA1, (M/4)*N * sizeof(double));
    hipMalloc((void **)&arrCUDA2, (M/4)*N * sizeof(double));
    hipMalloc((void **)&arrCUDA3, (M/4)*N * sizeof(double));
    hipMalloc((void **)&arrCUDA4, (M/4)*N * sizeof(double));

    hipEventCreate(&ct1);
    hipEventCreate(&ct2);
    hipEventRecord(ct1);
    //stream 1
    hipMemcpyAsync(&arrCUDA1, &arrP3,(M/4)*N*sizeof(double), hipMemcpyDeviceToHost, stream1);
    preguntaTres<<<gs, BS, 0, stream1>>>(arrCUDA1, dx);
    hipMemcpyAsync(&arrP3[0], &arrCUDA1[0], (M/4)*N*sizeof(double), hipMemcpyHostToDevice, stream1); 

    //stream 2
    hipMemcpyAsync(&arrCUDA2, &arrP3[M/4], (M/4)*N*sizeof(double), hipMemcpyDeviceToHost, stream2);
    preguntaTres<<<gs, BS, 0, stream2>>>(arrCUDA2, dx);
    hipMemcpyAsync(&arrP3[M*N/4], &arrCUDA2, (M/4)*N*sizeof(double), hipMemcpyHostToDevice, stream2);

    //stream 3
    hipMemcpyAsync(&arrCUDA3, &arrP3[M/2], (M/4)*N*sizeof(double), hipMemcpyDeviceToHost, stream3);
    preguntaTres<<<gs, BS, 0, stream3>>>(arrCUDA3, dx);
    hipMemcpyAsync(&arrP3[M*N/2], &arrCUDA3, (M/4)*N*sizeof(double), hipMemcpyHostToDevice, stream3);

    //stream 4
    hipMemcpyAsync(&arrCUDA4, &arrP3[3*M/4], (M/4)*N*sizeof(double), hipMemcpyDeviceToHost, stream4);
    preguntaTres<<<gs, BS, 0, stream4>>>(arrCUDA4, dx);
    hipMemcpyAsync(&arrP3[3*M*N/4], &arrCUDA4, (M/4)*N*sizeof(double), hipMemcpyHostToDevice, stream4);

    hipEventRecord(ct2);
    hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt, ct1, ct2);
	printf("3. Tiempo GPU: %f[ms]\n", dt);

    for(int i = 0; i < 20; i++){
        cout << arrP3[i] << " ";
    }
    cout << endl;

}