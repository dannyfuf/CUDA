#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <string>
#include <sys/types.h>
using namespace std;

#define BS 256
#define X 3
#define Y 3
#define cantStream 6
#define PYTHON_COMMAND 0 // 0 si el comando de python NO lleva el 3. cualquier otro valor si lo lleva.

//
// Funcion que ejecuta el comando de python para convertir la imagen en formato txt
// en formato txt a PNG.
// Hay que cambiar PYTHON_COMMAND dependiendo de como se ejecuta el comando en el PC.
//
int TXTtoRGB(){
    if (PYTHON_COMMAND == 0){
        cout << "Reconstruyendo Imagen..." << endl;
        system("python TXTtoRGB.py");
        return 1;
    }
    else if(system("python3 TXTtoRGB.py")){
        cout << "Reconstruyendo Imagen..." << endl;
        return 1;
    }
    else{
        cout << "Error al convertir el txt a imagen" << endl;
        return 0;
    }
}

//
// Funcion que imprime el ultimo error arrojado por CUDA
//
void cudaCheckError(int i) {
    hipError_t e=hipGetLastError();
    if(e!=hipSuccess) {
        printf("%d.- Cuda failure %s:%d: '%s'\n", i,__FILE__,__LINE__,hipGetErrorString(e));
        exit(0);
    }
}

/*
 *  Escritura Archivo txt
    Funcion extraida de actividad de curso
 */
void Write(float* R, float* G, float* B, 
	       int M, int N, const char *filename) {
    FILE *fp;
    fp = fopen(filename, "w");
    fprintf(fp, "%d %d\n", M, N);
    for(int i = 0; i < M*N-1; i++)
        fprintf(fp, "%f ", R[i]);
    fprintf(fp, "%f\n", R[M*N-1]);
    for(int i = 0; i < M*N-1; i++)
        fprintf(fp, "%f ", G[i]);
    fprintf(fp, "%f\n", G[M*N-1]);
    for(int i = 0; i < M*N-1; i++)
        fprintf(fp, "%f ", B[i]);
    fprintf(fp, "%f\n", B[M*N-1]);
    fclose(fp);
}

/*
 *  Lectura Archivo txt
 */
void Read(float** R, float** G, float** B, int *M, int *N, 
	      const char *filename) {    
    FILE *fp;
    fp = fopen(filename, "r");
    fscanf(fp, "%d %d\n", M, N);

    int imsize = (*M) * (*N);
    float* R1 = new float[imsize];
    float* G1 = new float[imsize];
    float* B1 = new float[imsize];

    for(int i = 0; i < imsize; i++)
        fscanf(fp, "%f ", &(R1[i]));
    for(int i = 0; i < imsize; i++)
        fscanf(fp, "%f ", &(G1[i]));
    for(int i = 0; i < imsize; i++)
        fscanf(fp, "%f ", &(B1[i]));
    
    fclose(fp);
    *R = R1; *G = G1; *B = B1;
}

__global__ void kernelStream(float* R, float* Rx, float *Ry, int M, int N, int Mout, int Nout, int *k1, int *k2, int tam){
    int tid = threadIdx.x + blockDim.x * blockIdx.x;// 
	if (tid < tam){
        float v1 = 0, v2 = 0;
        int fila = tid + (tid/Nout)*2;
        for(int i = 0; i<Y ; i++){
            for(int j = 0; j<X ; j++){
                v1 += R[j+i*N+fila]*k1[j+i*Y];
                v2 += R[j+i*N+fila]*k2[j+i*Y];
            }
        }
        Rx[tid] = v1;
        Ry[tid] = v2;
    }
}


__global__ void kernelConvolucion(float* R, float* Rx, float *Ry, int M, int N, int Mout, int Nout, int *k1, int *k2){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < Mout*Nout){
        float v1 = 0, v2 = 0;
        int fila = tid + (tid/Nout)*2;
        for(int i = 0; i<Y ; i++){
            for(int j = 0; j<X ; j++){
                v1 += R[j+i*N+fila]*k1[j+i*Y];
                v2 += R[j+i*N+fila]*k2[j+i*Y];
            }
        }
        Rx[tid] = v1;
        Ry[tid] = v2;
    }
}


__global__ void kernelFila(float* R, float* Rx, float *Ry, int M, int N, int Mout, int Nout, int *k1, int *k2){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < Mout){
        float v1, v2;
        for(int k=0; k<Nout ; k++){
            v1 = 0;
            v2 = 0;
            for(int i = 0; i<Y ; i++){
                for(int j = 0; j<X ; j++){
                    v1 += R[j+k+i*N+tid*N]*k1[j+i*Y];
                    v2 += R[j+k+i*N+tid*N]*k2[j+i*Y];
                }
            }
            Rx[k+tid*Nout] = v1;
            Ry[k+tid*Nout] = v2;
        }
    }
}

void kernelCPU(float* R, float* Rx, float *Ry, int M, int N, int Mout, int Nout, int *k1, int *k2){
    float v1, v2;
    for(int h=0 ; h<Mout ; h++){
        for(int k=0; k<Nout ; k++){
            v1 = 0;
            v2 = 0;
            for(int i = 0; i<Y ; i++){
                for(int j = 0; j<X ; j++){
                    v1 += R[j+k+i*N+h*N]*k1[j+i*Y];
                    v2 += R[j+k+i*N+h*N]*k2[j+i*Y];
                }
            }
            Rx[k+h*Nout] = v1;
            Ry[k+h*Nout] = v2;
        }
    }
}

void blancoynegro(float* R, float* G, float* B, int M, int N){
    float prom = 0;
    int imsize = (M) * (N);
    for(int i = 0; i < imsize; i++){
        prom = (R[i]+G[i]+B[i] )/ 3;
        (R[i]) = prom;
        (G[i]) = prom;
        (B[i]) = prom;
    }
}

void copiar(float *Rhost, float *Ghost, float *Bhost, float *Rxhost, float *Ryhost, int Mout, int Nout){
    float tmp;
    //norma
    for(int i = 0; i < Mout*Nout; i++){
        tmp = sqrt( (pow(Rxhost[i], 2)+ pow(Ryhost[i], 2)) );
        if(tmp > 1) tmp = 1;
        Rhost[i] = tmp;
        Ghost[i] = tmp;
        Bhost[i] = tmp;
    }

    Write(Rhost, Ghost, Bhost, Mout, Nout, "salida.txt");
}

void callKernelFila(float * Rhost, int N, int M, int Mout, int Nout, int * k1, int * k2){
    float *Rdev, *Rx, *Ry, *Rxhost, *Ryhost, dt;
    hipEvent_t ct1, ct2;
    int *k1dev, *k2dev;
    
    int GS = (int)ceil((float) Mout / BS);

    hipMalloc((void**)&Rdev, M * N * sizeof(float));
    //cudaMemcpy(Rdev, Rhost, M * N * sizeof(float), cudaMemcpyHostToDevice);
    hipMemcpy(Rdev, Rhost, M * N * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError(1);

    hipMalloc((void**)&k1dev, X * Y * sizeof(int));
    hipMalloc((void**)&k2dev, X * Y * sizeof(int));
    hipMemcpy(k1dev, k1, X * Y * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(k2dev, k2, X * Y * sizeof(int), hipMemcpyHostToDevice);
    cudaCheckError(2);

    hipMalloc((void**)&Rx, Mout * Nout * sizeof(float));
    hipMalloc((void**)&Ry, Mout * Nout * sizeof(float));
    cudaCheckError(3);

    hipEventCreate(&ct1);
    hipEventCreate(&ct2);
    hipEventRecord(ct1);

    kernelFila<<<GS, BS>>>(Rdev, Rx, Ry, M, N, Mout, Nout, k1dev, k2dev);
    // cudaCheckError(4);

    Rxhost = new float[Mout*Nout];
	Ryhost = new float[Mout*Nout];
    hipMemcpy(Rxhost, Rx, Mout * Nout * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(Ryhost, Ry, Mout * Nout * sizeof(float), hipMemcpyDeviceToHost);
    
    hipEventRecord(ct2);
    hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt, ct1, ct2);
    cout << "Tiempo GPU una hebra por fila: " << dt << "[ms]" << endl;
    // cudaCheckError(5);

    float *Rfinal= new float[Mout*Nout];
    float *Gfinal = new float[Mout*Nout];
    float *Bfinal = new float[Mout*Nout];
    copiar(Rfinal, Gfinal, Bfinal, Rxhost, Ryhost, Mout, Nout);

}

void callCPU(float * Rhost, int N, int M, int Mout, int Nout, int * k1, int * k2){
    clock_t t1, t2;
    double ms;
    t1 = clock();
    float *Rx = new float[Mout*Nout];
    float *Ry = new float[Mout*Nout];
    kernelCPU(Rhost, Rx, Ry, M, N, Mout, Nout, k1, k2);
    t2 = clock();
    ms = 1000.0 * (double)(t2 - t1) / CLOCKS_PER_SEC;
    cout << "Tiempo CPU es: "<< ms << "[ms]" << endl;

    float *Rfinal= new float[Mout*Nout];
    float *Gfinal = new float[Mout*Nout];
    float *Bfinal = new float[Mout*Nout];
    copiar(Rfinal, Gfinal, Bfinal, Rx, Ry, Mout, Nout);
}

void callKernelConv(float * Rhost, int N, int M, int Mout, int Nout, int * k1, int * k2){
    float *Rdev, *Rx, *Ry, *Rxhost, *Ryhost, dt;
    hipEvent_t ct1, ct2;
    int *k1dev, *k2dev;
    
    int GS = (int)ceil((float) Mout*Nout / BS);

    cudaCheckError(1);

    hipMalloc((void**)&k1dev, X * Y * sizeof(int));
    hipMalloc((void**)&k2dev, X * Y * sizeof(int));
    hipMemcpy(k1dev, k1, X * Y * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(k2dev, k2, X * Y * sizeof(int), hipMemcpyHostToDevice);
    cudaCheckError(2);

    hipMalloc((void**)&Rx, Mout * Nout * sizeof(float));
    hipMalloc((void**)&Ry, Mout * Nout * sizeof(float));
    cudaCheckError(3);

    hipEventCreate(&ct1);
    hipEventCreate(&ct2);
    hipEventRecord(ct1);
    
    hipMalloc((void**)&Rdev, M * N * sizeof(float));
    hipMemcpy(Rdev, Rhost, M * N * sizeof(float), hipMemcpyHostToDevice);

    kernelConvolucion<<<GS, BS>>>(Rdev, Rx, Ry, M, N, Mout, Nout, k1dev, k2dev);
    cudaCheckError(4);

    Rxhost = new float[Mout*Nout];
	Ryhost = new float[Mout*Nout];
    hipMemcpy(Rxhost, Rx, Mout * Nout * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(Ryhost, Ry, Mout * Nout * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError(5);

    hipEventRecord(ct2);
    hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt, ct1, ct2);
    cout << "Tiempo GPU una hebra por convolucion: " << dt << "[ms]" << endl;
    
    float *Rfinal= new float[Mout*Nout];
    float *Gfinal = new float[Mout*Nout];
    float *Bfinal = new float[Mout*Nout];
    copiar(Rfinal, Gfinal, Bfinal, Rxhost, Ryhost, Mout, Nout);
}

void callKernelStream(float * Rhost, int N, int M, int Mout, int Nout, int * k1, int * k2){
    hipStream_t stream1, stream2, stream3, stream4, stream5, stream6;
    float *Rdev, *Rxhost, *Ryhost, dt;
    float *RxStream1, *RxStream2, *RxStream3, *RxStream4, *RxStream5, *RxStream6;
    float *RyStream1, *RyStream2, *RyStream3, *RyStream4, *RyStream5, *RyStream6;
    hipEvent_t ct1, ct2;
    int *k1dev, *k2dev;

    Rxhost = new float[Mout*Nout];
    Ryhost = new float[Mout*Nout];

    // int cantStream = 4;
    int GS = (int)ceil((float) (Mout/cantStream)*Nout / BS);
    int GS4 = (int)ceil((float) ((Mout+Mout%cantStream)/cantStream)*Nout / BS);

    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    hipStreamCreate(&stream4);
    hipStreamCreate(&stream5);
    hipStreamCreate(&stream6);

    int size = (int)(Mout/cantStream)*Nout;
    int size4 = (int)((Mout/cantStream)+Mout%cantStream)*Nout;

    hipMalloc((void **)&RxStream1, size * sizeof(float));
    hipMalloc((void **)&RyStream1, size * sizeof(float));

    hipMalloc((void **)&RxStream2, size * sizeof(float));
    hipMalloc((void **)&RyStream2, size * sizeof(float));
    
    hipMalloc((void **)&RxStream3, size * sizeof(float));
    hipMalloc((void **)&RyStream3, size * sizeof(float));

    hipMalloc((void **)&RxStream4, size * sizeof(float));
    hipMalloc((void **)&RyStream4, size * sizeof(float));

    hipMalloc((void **)&RxStream5, size * sizeof(float));
    hipMalloc((void **)&RyStream5, size * sizeof(float));

    hipMalloc((void **)&RxStream6, size4 * sizeof(float));
    hipMalloc((void **)&RyStream6, size4 * sizeof(float));
    


    // copiar el kernel de convolucion a memoria de gpu
    hipMalloc((void **)&k1dev, X * Y * sizeof(float));
    hipMalloc((void **)&k2dev, X * Y * sizeof(float));
    hipMemcpy(k1dev, k1, X * Y * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(k2dev, k2, X * Y * sizeof(float), hipMemcpyHostToDevice);
    
    float * RH;
    hipHostMalloc((void **)&RH, M * N * sizeof(float), hipHostMallocDefault); //para copiar a los streams de forma eficiente

    for(int i = 0; i < M*N; i++){
        RH[i] = Rhost[i];
    }

    int sizeFull = (int)(M/cantStream)*N;
    int sizeFull6 = (int)((M/cantStream)+M%cantStream)*N;
    
    float *Rdev2, *Rdev3, *Rdev4, *Rdev5, *Rdev6;
    hipMalloc((void **)&Rdev, (sizeFull+2*N) * sizeof(float));
    hipMalloc((void **)&Rdev2, (sizeFull+2*N) * sizeof(float));
    hipMalloc((void **)&Rdev3, (sizeFull+2*N) * sizeof(float));
    hipMalloc((void **)&Rdev4, (sizeFull+2*N) * sizeof(float));
    hipMalloc((void **)&Rdev5, (sizeFull+2*N) * sizeof(float));
    hipMalloc((void **)&Rdev6, (sizeFull6+2*N) * sizeof(float));

    hipEventCreate(&ct1);
    hipEventCreate(&ct2);
    hipEventRecord(ct1);
    // copiar la imagen completa a memoria de gpu
    hipMemcpyAsync(Rdev, &RH[0],(sizeFull+2*N) * sizeof(float), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(Rdev2, &RH[sizeFull], (sizeFull+2*N) * sizeof(float), hipMemcpyHostToDevice, stream2);
    hipMemcpyAsync(Rdev3, &RH[2*sizeFull], (sizeFull+2*N) * sizeof(float), hipMemcpyHostToDevice, stream3);
    hipMemcpyAsync(Rdev4, &RH[3*sizeFull], (sizeFull+2*N) * sizeof(float), hipMemcpyHostToDevice, stream4);
    hipMemcpyAsync(Rdev5, &RH[4*sizeFull], (sizeFull+2*N) * sizeof(float), hipMemcpyHostToDevice, stream5);
    hipMemcpyAsync(Rdev6, &RH[5*sizeFull], (sizeFull6) * sizeof(float), hipMemcpyHostToDevice, stream6);

    //stream 1
    kernelStream<<<GS, BS, 0, stream1>>>(Rdev, RxStream1, RyStream1, M, N, Mout, Nout, k1dev, k2dev, size);
    hipMemcpyAsync(&Rxhost[0], RxStream1, size*sizeof(float), hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(&Ryhost[0], RyStream1, size*sizeof(float), hipMemcpyDeviceToHost, stream1);

    //stream 2
    kernelStream<<<GS, BS, 0, stream2>>>(Rdev2, RxStream2, RyStream2, M, N, Mout, Nout, k1dev, k2dev, size);
    hipMemcpyAsync(&Rxhost[size], RxStream2, size*sizeof(float), hipMemcpyDeviceToHost, stream2);
    hipMemcpyAsync(&Ryhost[size], RyStream2, size*sizeof(float), hipMemcpyDeviceToHost, stream2);

    //stream 3
    kernelStream<<<GS, BS, 0, stream3>>>(Rdev3, RxStream3, RyStream3, M, N, Mout, Nout, k1dev, k2dev, size);
    hipMemcpyAsync(&Rxhost[size*2], RxStream3, size*sizeof(float), hipMemcpyDeviceToHost, stream3);
    hipMemcpyAsync(&Ryhost[size*2], RyStream3, size*sizeof(float), hipMemcpyDeviceToHost, stream3);
    
    //stream 4
    kernelStream<<<GS, BS, 0, stream4>>>(Rdev4, RxStream4, RyStream4, M, N, Mout, Nout, k1dev, k2dev, size);
    hipMemcpyAsync(&Rxhost[size*3], RxStream4, size*sizeof(float), hipMemcpyDeviceToHost, stream4);
    hipMemcpyAsync(&Ryhost[size*3], RyStream4, size*sizeof(float), hipMemcpyDeviceToHost, stream4);
    //stream 5
    kernelStream<<<GS, BS, 0, stream5>>>(Rdev5, RxStream5, RyStream5, M, N, Mout, Nout, k1dev, k2dev, size);
    hipMemcpyAsync(&Rxhost[size*4], RxStream5, size*sizeof(float), hipMemcpyDeviceToHost, stream5);
    hipMemcpyAsync(&Ryhost[size*4], RyStream5, size*sizeof(float), hipMemcpyDeviceToHost, stream5);

    //stream 6
    kernelStream<<<GS4, BS, 0, stream6>>>(Rdev6, RxStream6, RyStream6, M, N, Mout, Nout, k1dev, k2dev, size4);
    hipMemcpyAsync(&Rxhost[size*5], RxStream6, size4*sizeof(float), hipMemcpyDeviceToHost, stream6);
    hipMemcpyAsync(&Ryhost[size*5], RyStream6, size4*sizeof(float), hipMemcpyDeviceToHost, stream6);

    hipEventRecord(ct2);
    hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt, ct1, ct2);
	printf("Tiempo GPU Streams: %f[ms]\n", dt);

    hipDeviceSynchronize();


    float *Rfinal= new float[Mout*Nout];
    float *Gfinal = new float[Mout*Nout];
    float *Bfinal = new float[Mout*Nout];
    copiar(Rfinal, Gfinal, Bfinal, Rxhost, Ryhost, Mout, Nout);
    
}

int main(){

    //inicializacion
    //se convierte la imagen a blanco y negro
    float *Rhost, *Ghost, *Bhost;
    int M, N, Mout, Nout; //M filas, N columnas
    Read(&Rhost, &Ghost, &Bhost, &M, &N, "mono.txt"); 
    blancoynegro(Rhost, Ghost, Bhost, M, N);
    Nout = N - 2;
	Mout = M - 2;
    //probar esto
    
    int *k1{ new int[9]{ -1, 0, 1, -2, 0, 2, -1, 0, 1 } };
    int *k2{ new int[9]{ -1, -2, -1, 0, 0, 0, 1, 2, 1 } };

    // llamada a la implementacion de cpu
    // callCPU(Rhost, N, M, Mout, Nout, k1, k2);
    // TXTtoRGB();

    // llamada a la implementación del kernel usando una hebra por fila. 
    //callKernelFila(Rhost, N, M, Mout, Nout, k1, k2);
    //TXTtoRGB();

    // llamada a la implementación del kernel usando una hebra por fila. 
    callKernelConv(Rhost, N, M, Mout, Nout, k1, k2);
    //TXTtoRGB();

    // llamada al kernel usando streams y un kernel por calculo.
    callKernelStream(Rhost, N, M, Mout, Nout, k1, k2);
    TXTtoRGB();

    return 0;
}